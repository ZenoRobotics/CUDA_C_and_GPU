#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

//#define CUBLAS 1

// System includes
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <helper_string.h>  // helper for shared functions common to CUDA SDK samples
#include <hip/hip_runtime_api.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#ifdef CUBLAS
   #include <hipblas.h>
#endif

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif


typedef struct _matrixSize      // Optional Command-line multiplier for matrix sizes
{
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
} sMatrixSize;

const int BLOCK_SIZE = 32;

#ifndef CUBLAS
/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
__global__ void
matrixMulCUDA(float *C, float *A, float *B, unsigned int wA, unsigned int wB)
{   
    // Block dimensions
    int bDimx = blockDim.x;
    int bDimy = blockDim.y;

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * bDimy * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = bDimx; //BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = bx * bDimx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = bDimx * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];
        
        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll 

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * bDimy * by + bDimx * bx;
    C[c + wB * ty + tx] = Csub;
    
}
#endif

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void
matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
        {
            float sum = 0;

            for (unsigned int k = 0; k < wA; ++k)
            {
                float a = A[i * wA + k];
                float b = B[k * wB + j];
                sum += a * b;
            }

            C[i * wB + j] = (float)sum;
        }
}

// Allocates a matrix with random float entries.
void randomInit(float *data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;

    for (j = 0; j < height; j++)
    {
        if (error_count < iListLength)
        {
            printf("\n  Row %d:\n", j);
        }

        for (i = 0; i < width; i++)
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);

            if (fDiff > fListTol)
            {
                if (error_count < iListLength)
                {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }

                error_count++;
            }
        }
    }

    printf(" \n  Total Errors = %d\n", error_count);
}

void initializeCUDA(int argc, char **argv, int &devID, int &iSizeMultiple, sMatrixSize &matrix_size)
{
    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    hipError_t error;
    devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        error = hipSetDevice(devID);

        if (error != hipSuccess)
        {
            printf("hipSetDevice returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
    }

    // get number of SMs on this GPU
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }


    if (checkCmdLineFlag(argc, (const char **)argv, "sizemult"))
    {
        iSizeMultiple = getCmdLineArgumentInt(argc, (const char **)argv, "sizemult");
    }

    iSizeMultiple = min(iSizeMultiple, 10);
    iSizeMultiple = max(iSizeMultiple, 1);

    hipDeviceProp_t deviceProp;

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

    // use a larger block size for Fermi and above
    //int block_size = (deviceProp.major < 2) ? 16 : 32;

    matrix_size.uiWA = 1024; //4 * block_size * iSizeMultiple;
    matrix_size.uiHA = 1024; //4 * block_size * iSizeMultiple;
    matrix_size.uiWB = 1024; //4 * block_size * iSizeMultiple;
    matrix_size.uiHB = 1024; //4 * block_size * iSizeMultiple;
    matrix_size.uiWC = matrix_size.uiWB;
    matrix_size.uiHC = matrix_size.uiHA;

    printf("MatrixA(%u,%u), MatrixB(%u,%u), MatrixC(%u,%u)\n",
           matrix_size.uiWA, matrix_size.uiHA,
           matrix_size.uiWB, matrix_size.uiHB,
           matrix_size.uiWC, matrix_size.uiHC);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test matrix multiply using CUBLAS
////////////////////////////////////////////////////////////////////////////////
int matrixMultiply(int argc, char **argv, int devID, sMatrixSize &matrix_size)
{
    hipDeviceProp_t deviceProp;
    hipError_t error;

    // execute the kernel
    int nIter = 30;

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // use a larger block size for Fermi and above
    //int block_size = 32; //(deviceProp.major < 2) ? 16 : 32;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = matrix_size.uiWA * matrix_size.uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = matrix_size.uiWB * matrix_size.uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // set seed for rand()
    srand(2006);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float *h_C      = (float *) malloc(mem_size_C);
    float *h_CUBLAS = (float *) malloc(mem_size_C);

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_A h_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_B h_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // setup execution parameters
    //dim3 threads(block_size, block_size);
    //dim3 grid(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);

    // Setup execution parameters
    int  wBlock = BLOCK_SIZE;  // Value may change based on dimsB.x (wB)
    int  hBlock = BLOCK_SIZE;  // Value may change based on wBlock and dimsA.x (wA) 
    //bool Done   = false;
/*
    if ((matrix_size.uiWB > 1) && ((matrix_size.uiWB%2) == 0)) 
    {
       while (!Done)
       {
          if ((matrix_size.uiWB%wBlock) > 0)
             wBlock = wBlock/2;
          else
             Done = true;
       }
       hBlock = 1024/wBlock;
    }
    else if (matrix_size.uiWB == 1)
    {
       wBlock = 1;
       hBlock = matrix_size.uiHB;  // Good for up to 1024
    }
    else
    {
       printf("Width of B Matrix is not a power of 2.\n");
       exit(EXIT_FAILURE);
    }
*/
    dim3 threads(wBlock, hBlock);                                             // (x = cols, y = rows)
    dim3 grid(matrix_size.uiWB / threads.x, matrix_size.uiHA / threads.y);    // (x = cols, y = rows)
    
    // Debug
    printf("\n");
    printf("hBlock = %d, wBlock = %d, hGrid = %d, wGrid = %d\n\n", hBlock, wBlock, grid.y, grid.x);

    #ifdef CUBLAS
       // create and start timer
       printf("Computing result using CUBLAS...");
    #else
       printf("Computing result using Homebrew Kernel...");
    #endif


    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
         fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
         exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
         fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
         exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
         fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
         exit(EXIT_FAILURE);
    }

    #ifdef CUBLAS
        hipblasHandle_t handle;

        hipblasStatus_t ret;

        ret = hipblasCreate(&handle);

        if (ret != HIPBLAS_STATUS_SUCCESS)
        {
            printf("hipblasCreate returned error code %d, line(%d)\n", ret, __LINE__);
            exit(EXIT_FAILURE);
        }

        const float alpha = 1.0f;
        const float beta  = 0.0f;

    #endif
hipProfilerStart();

        for (int j = 0; j < nIter; j++)
        {
            //note cublas is column primary!
            //need to transpose the order

            #ifdef CUBLAS
            ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, 
                             &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWA);
            
               if (ret != HIPBLAS_STATUS_SUCCESS)
               {
                  printf("hipblasSgemm returned error code %d, line(%d)\n", ret, __LINE__);
                  exit(EXIT_FAILURE);
               }
            
            #else
               matrixMulCUDA<<< grid, threads >>>(d_C, d_A, d_B, matrix_size.uiWA, matrix_size.uiWB);
            #endif
            
        }
hipProfilerStop();

     printf("done.\n");

     // Record the stop event
     error = hipEventRecord(stop, NULL);

     if (error != hipSuccess)
     {
         fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
         exit(EXIT_FAILURE);
     }

     // Wait for the stop event to complete
     error = hipEventSynchronize(stop);

     if (error != hipSuccess)
     {
         fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
         exit(EXIT_FAILURE);
     }

     float msecTotal = 0.0f;
     error = hipEventElapsedTime(&msecTotal, start, stop);

     if (error != hipSuccess)
     {
         fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
         exit(EXIT_FAILURE);
     }

     // Compute and print the performance
     float msecPerMatrixMul = msecTotal / nIter;
     double flopsPerMatrixMul = ((double)matrix_size.uiWA + ((double)matrix_size.uiWA - 1)) * (double)matrix_size.uiHA * (double)matrix_size.uiWB;
     double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
     printf("\n");
     printf(
         "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n\n",
         gigaFlops,
         msecPerMatrixMul,
         flopsPerMatrixMul);

     // copy result from device to host
     error = hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost);

     if (error != hipSuccess)
     {
         printf("hipMemcpy h_CUBLAS d_C returned error code %d, line(%d)\n", error, __LINE__);
         exit(EXIT_FAILURE);
     }


    // compute reference solution
    printf("Computing result using host CPU...");
    float *reference = (float *)malloc(mem_size_C);
    clock_t tStart = clock();
    matrixMulCPU(reference, h_A, h_B, matrix_size.uiHA, matrix_size.uiWA, matrix_size.uiWB);
    clock_t tStop = clock();
    printf("done.\n");
    printf("Time taken for CPU version: %.2fs\n\n", (double)(tStop - tStart)/CLOCKS_PER_SEC);

    // check result (CUBLAS)
    bool resCUBLAS = sdkCompareL2fe(reference, h_CUBLAS, size_C, 1.0e-6f);

    if (resCUBLAS != true)
    {
        printDiff(reference, h_CUBLAS, matrix_size.uiWC, matrix_size.uiHC, 100, 1.0e-5f);
    }

    printf("Comparing CUBLAS Matrix Multiply with CPU results: %s\n", (true == resCUBLAS) ? "OK" : "FAIL");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    if (resCUBLAS == true)
    {
        return EXIT_SUCCESS;    // return value = 1
    }
    else
    {
        return EXIT_FAILURE;     // return value = 0
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("[Matrix Multiply] - Starting...\n");

    int devID = 0, sizeMult = 5;
    sMatrixSize matrix_size;

    initializeCUDA(argc, argv, devID, sizeMult, matrix_size);

    int matrix_result = matrixMultiply(argc, argv, devID, matrix_size);

    exit(matrix_result);
}


